#include "hip/hip_runtime.h"
#include "activations/ClippedReLU.h"
#include "activations/Linear.h"
#include "activations/ReLU.h"
#include "activations/Sigmoid.h"
#include "data/DenseMatrix.h"
#include "data/SArray.h"
#include "data/Tape.h"
#include "dataset/batchloader.h"
#include "dataset/dataset.h"
#include "dataset/io.h"
#include "dataset/reader.h"
#include "dataset/writer.h"
#include "dataset/shuffle.h"
#include "layer/DenseLayer.h"
#include "layer/DuplicateDenseLayer.h"
#include "loss/MLE.h"
#include "loss/MPE.h"
#include "loss/MSE.h"
#include "mappings.h"
#include "misc/csv.h"
#include "misc/timer.h"
#include "network/Network.h"
#include "operations/operations.h"
#include "optimizer/Adam.h"
#include "position/fenparsing.h"
#include "position/position.h"
#include "quantitize.h"

#include <filesystem>
#include <iostream>

const std::string data_path = "E:/berserk/training-data/berserk9dev2/finny-data/";
std::string output = "./resources/runs/exp4/";

int main() {
    init();

    // definitions
    constexpr uint32_t       I = 8 * 12 * 64;
    constexpr uint32_t      L1 = 256;
    constexpr uint32_t      L2 = 32;
    constexpr uint32_t      L3 = 32;  
    constexpr uint32_t       O = 1;
    constexpr uint32_t       B = 8192;
    constexpr uint32_t     BPE = 100000000 / B;
    constexpr  int32_t       E = 600;

    // Load files
    std::vector<std::string> files {};
    for (int i = 0; i < 7; i++)
        files.push_back(data_path + "berserk9dev2.d9." + std::to_string(i) + ".bin");

    BatchLoader  batch_loader {files, B};

    // Input data (perspective)
    SparseInput  i0 {I, B, 32};    // 32 max inputs
    SparseInput  i1 {I, B, 32};

    DenseMatrix  target {O, B};
    SArray<bool> target_mask {O * B};
    target_mask.malloc_cpu();
    target_mask.malloc_gpu();

    const float QUANT_ONE = 127.0;
    DuplicateDenseLayer<I, L1, ClippedReLU> l1 {};
    dynamic_cast<ClippedReLU*>(l1.getActivationFunction())->max = 1.0;

    const float SCALE_HIDDEN = 64.0;
    DenseLayer<2 * L1, L2, ClippedReLU> l2 {};
    dynamic_cast<ClippedReLU*>(l2.getActivationFunction())->max = 1.0;
    l2.getTunableParameters()[0]->min_allowed_value = -QUANT_ONE / SCALE_HIDDEN;
    l2.getTunableParameters()[0]->max_allowed_value = QUANT_ONE / SCALE_HIDDEN;

    DenseLayer<L2, L3, ClippedReLU> l3 {};
    dynamic_cast<ClippedReLU*>(l3.getActivationFunction())->max = 1.0;
    l3.getTunableParameters()[0]->min_allowed_value = -QUANT_ONE / SCALE_HIDDEN;
    l3.getTunableParameters()[0]->max_allowed_value = QUANT_ONE / SCALE_HIDDEN;

    const float SCALE_OUT = 16.0;
    const float NN_SCALE = 231.0;
    DenseLayer<L3, O, Sigmoid> l4 {};
    dynamic_cast<Sigmoid*>(l4.getActivationFunction())->scalar = NN_SCALE / 139;
    l4.getTunableParameters()[0]->min_allowed_value = -(QUANT_ONE * QUANT_ONE) / (SCALE_OUT * NN_SCALE);
    l4.getTunableParameters()[0]->max_allowed_value = (QUANT_ONE * QUANT_ONE) / (SCALE_OUT * NN_SCALE);

    // stack layers to build network
    std::vector<LayerInterface*> layers {};
    layers.push_back(&l1);
    layers.push_back(&l2);
    layers.push_back(&l3);
    layers.push_back(&l4);

    Network network {layers};

    // loss function
    MPE     loss_function {2.5, true};
    network.setLossFunction(&loss_function);

    // optimizer
    Adam adam {};
    adam.init(layers);
    adam.alpha = 0.001;
    adam.beta1 = 0.9;
    adam.beta2 = 0.999;
    adam.eps = 1e-7;

    CSVWriter csv {output + "loss.csv"};

    Timer t {};
    for (int epoch = 1; epoch <= E; epoch++) {
        float epoch_loss = 0;
        long long prev_duration = 0;

        t.tick();

        for (int batch = 1; batch <= BPE; batch++) {
            // get the next dataset (batch)
            auto* ds = batch_loader.next();
            // assign to the inputs and compute the target
            dense_berky::assign_inputs_batch(*ds, i0, i1, target, target_mask);
            // upload relevant data
            i0.column_indices.gpu_upload();
            i1.column_indices.gpu_upload();
            target.gpu_upload();
            target_mask.gpu_upload();

            // download the loss to display the loss of the iteration
            loss_function.loss.gpu_download();

            // measure time and print output
            t.tock();
            if (batch == BPE || t.duration() - prev_duration > 1000) {
                prev_duration = t.duration();

                std::printf("\rep/ba = [%3d/%5d], ", epoch, batch + 1);
                std::printf("batch_loss = [%1.8f], ", loss_function.loss(0));
                std::printf("epoch_loss = [%1.8f], ", epoch_loss / (batch + 1));
                std::printf("speed = [%9d pos/s], ", (int) std::round(1000.0f * B * (batch + 1) / t.duration()));
                std::printf("time = [%3ds]", (int) t.duration() / 1000);
                std::cout << std::flush;
            }

            epoch_loss += loss_function.loss(0);
            // make sure to reset the loss here since the mse increments the loss in order to not have
            // to use memcpy (might change soon)
            loss_function.loss(0) = 0;
            loss_function.loss.gpu_upload();

            // feed forward
            network.batch(std::vector<SparseInput*> {&i0, &i1}, target, target_mask);

            // update weights
            adam.apply(1);
        }

        std::cout << std::endl;

        csv.write({std::to_string(epoch),  std::to_string(epoch_loss / BPE)});
        write_4(output + "nn-epoch" + std::to_string(epoch) + ".nnue", network, QUANT_ONE, SCALE_HIDDEN, SCALE_OUT, NN_SCALE);
        
        if (epoch % 100 == 0)
            adam.alpha *= 0.3;
    }

    close();
}
