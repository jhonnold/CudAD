#include "hip/hip_runtime.h"
#include "activations/ClippedReLU.h"
#include "activations/Linear.h"
#include "activations/ReLU.h"
#include "activations/Sigmoid.h"
#include "data/DenseMatrix.h"
#include "data/SArray.h"
#include "data/Tape.h"
#include "dataset/batchloader.h"
#include "dataset/dataset.h"
#include "dataset/io.h"
#include "dataset/reader.h"
#include "dataset/writer.h"
#include "dataset/shuffle.h"
#include "layer/DenseLayer.h"
#include "layer/DuplicateDenseLayer.h"
#include "loss/MLE.h"
#include "loss/MPE.h"
#include "loss/MSE.h"
#include "mappings.h"
#include "misc/csv.h"
#include "misc/timer.h"
#include "network/Network.h"
#include "operations/operations.h"
#include "optimizer/Adam.h"
#include "position/fenparsing.h"
#include "position/position.h"
#include "quantitize.h"

#include <filesystem>
#include <iostream>

const std::string data_path = "E:/berserk/training-data/master/";
std::string output = "./resources/runs/exp11/";

int main() {
    init();
    
    // definitions
    constexpr uint32_t       I = 8 * 12 * 64;
    constexpr uint32_t      L1 = 512;
    constexpr uint32_t      L2 = 32;
    constexpr uint32_t      L3 = 32;
    constexpr uint32_t       O = 1;
    constexpr uint32_t       B = 16384;
    constexpr uint32_t     BPE = 100000000 / B;
    constexpr  int32_t       E = 600;

    // Load files
    std::vector<std::string> files {};
    for (int i = 0; i < 10; i++)
        files.push_back(data_path + "berserk9dev2.d9." + std::to_string(i) + ".bin");

    BatchLoader  batch_loader {files, B};

    // Input data (perspective)
    SparseInput  i0 {I, B, 32};    // 32 max inputs
    SparseInput  i1 {I, B, 32};

    DenseMatrix  target {O, B};
    SArray<bool> target_mask {O * B};
    target_mask.malloc_cpu();
    target_mask.malloc_gpu();

    DuplicateDenseLayer<I, L1, ReLU> l1 {};
    DenseLayer<2 * L1, L2, ReLU> l2 {};
    DenseLayer<L2, L3, ReLU> l3 {};
    DenseLayer<L3, O, Sigmoid> l4 {};
    dynamic_cast<Sigmoid*>(l4.getActivationFunction())->scalar = 1 / 139.0;

    // stack layers to build network
    std::vector<LayerInterface*> layers {};
    layers.push_back(&l1);
    layers.push_back(&l2);
    layers.push_back(&l3);
    layers.push_back(&l4);

    Network network {layers};

    // loss function
    MPE     loss_function {2.5, false};
    network.setLossFunction(&loss_function);

    // optimizer
    Adam adam {};
    adam.init(layers);
    adam.alpha = 0.01;
    adam.beta1 = 0.95;
    adam.beta2 = 0.999;

    CSVWriter csv {output + "loss.csv"};

    Timer t {};
    for (int epoch = 1; epoch <= E; epoch++) {
        float epoch_loss = 0;
        long long prev_duration = 0;

        t.tick();

        for (int batch = 1; batch <= BPE; batch++) {
            // get the next dataset (batch)
            auto* ds = batch_loader.next();
            // assign to the inputs and compute the target
            dense_berky::assign_inputs_batch(*ds, i0, i1, target, target_mask);
            // upload relevant data
            i0.column_indices.gpu_upload();
            i1.column_indices.gpu_upload();
            target.gpu_upload();
            target_mask.gpu_upload();

            // download the loss to display the loss of the iteration
            loss_function.loss.gpu_download();

            // measure time and print output
            t.tock();
            if (batch == BPE || t.duration() - prev_duration > 1000) {
                prev_duration = t.duration();

                std::printf("\rep/ba = [%3d/%5d], ", epoch, batch + 1);
                std::printf("batch_loss = [%1.8f], ", loss_function.loss(0));
                std::printf("epoch_loss = [%1.8f], ", epoch_loss / (batch + 1));
                std::printf("speed = [%9d pos/s], ", (int) std::round(1000.0f * B * (batch + 1) / t.duration()));
                std::printf("time = [%3ds]", (int) t.duration() / 1000);
                std::cout << std::flush;
            }

            epoch_loss += loss_function.loss(0);
            // make sure to reset the loss here since the mse increments the loss in order to not have
            // to use memcpy (might change soon)
            loss_function.loss(0) = 0;
            loss_function.loss.gpu_upload();

            // feed forward
            network.batch(std::vector<SparseInput*> {&i0, &i1}, target, target_mask);

            // update weights
            adam.apply(1);
        }

        std::cout << std::endl;

        csv.write({std::to_string(epoch),  std::to_string(epoch_loss / BPE)});
        write_4(output + "nn-epoch" + std::to_string(epoch) + ".nnue", network);

        if (epoch % 100 == 0)
            adam.alpha *= 0.3;
    }

    close();
}
