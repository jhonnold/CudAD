#include "hip/hip_runtime.h"
/**
    CudAD is a CUDA neural network trainer, specific for chess engines.
    Copyright (C) 2022 Finn Eggers

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <https://www.gnu.org/licenses/>.
 */

#include "archs/Berserk.h"
#include "misc/config.h"
#include "trainer.h"
#include "data/DenseMatrix.h"
#include "data/SArray.h"
#include "data/SparseInput.h"
#include "dataset/batchloader.h"
#include "dataset/dataset.h"
#include "loss/Loss.h"
#include "misc/csv.h"
#include "misc/timer.h"
#include "network/Network.h"
#include "optimizer/Optimiser.h"
#include "quantitize.h"
#include "dataset/writer.h"

#include <iostream>
#include <vector>

constexpr int BatchSize = 16384;
constexpr int BatchesPerFile = 6103;
constexpr int TotalFiles = 74;

using namespace std;

int main() {
    init();

    const string data_path = "E:/berserk/training-data/master/";
    const string output    = "./resources/runs/exp100/";
    vector<string> files {};
    for (int i = 0; i < 20; i++)
        files.push_back(data_path + "n5k." + to_string(i) + ".bin");
    
    BatchLoader batch_loader {files, BatchSize};
    batch_loader.start();


    tuple<SparseInput, SparseInput> inputs {SparseInput {Berserk::Inputs, BatchSize, 32},
                                            SparseInput {Berserk::Inputs, BatchSize, 32}};
    DenseMatrix                     target {Berserk::Outputs, BatchSize};
    SArray<bool>                    target_mask {Berserk::Outputs * BatchSize};

    target_mask.malloc_cpu();
    target_mask.malloc_gpu();

    vector<LayerInterface*> layers = Berserk::get_layers();
    Network* network = new Network(layers);
    network->setLossFunction(Berserk::get_loss_function());
    network->loadWeights("./resources/runs/exp106/weights-epoch600.nnue");

    std::cout << "Loaded network" << std::endl;

    DataSet data_to_write {};

    uint64_t batch_num = 0;
    while (batch_num++ < BatchesPerFile * TotalFiles) {
        printf("Running batch %lld\n", batch_num);

        auto* ds = batch_loader.next();

        Berserk::assign_inputs_batch(*ds, get<0>(inputs), get<1>(inputs), target, target_mask);

        get<0>(inputs).column_indices.gpu_upload();
        get<1>(inputs).column_indices.gpu_upload();
        target.gpu_upload();
        target_mask.gpu_upload();

        network->feed(vector<SparseInput*> {&get<0>(inputs), &get<1>(inputs)});

        auto* values = &network->getOutput().values;
        values->gpu_download();

        for (size_t i = 0; i < BatchSize; i++) {
            auto eval = (int16_t) round(values->get(i));
            auto* pos = &ds->positions[i];

            if (pos->m_meta.getActivePlayer() == BLACK)
                eval = -eval;

            pos->m_result.score = eval;
        }

        data_to_write.addData(*ds);

        if (batch_num % BatchesPerFile == 0) {
            int idx = batch_num / BatchesPerFile;

            write("E:/berserk/training-data/rescored/n5k." + to_string(idx) + ".bin", data_to_write);

            data_to_write.clear();
        }
    }

    // Trainer<Berserk, 600> trainer {};
    // trainer.fit(files, vector<string> {data_path + "validation.bin"}, output);

    close();
}
