#include "hip/hip_runtime.h"
#include "activations/ClippedReLU.h"
#include "activations/Linear.h"
#include "activations/ReLU.h"
#include "activations/Sigmoid.h"
#include "data/DenseMatrix.h"
#include "data/SArray.h"
#include "data/Tape.h"
#include "dataset/batchloader.h"
#include "dataset/dataset.h"
#include "dataset/io.h"
#include "dataset/reader.h"
#include "dataset/writer.h"
#include "dataset/shuffle.h"
#include "layer/DenseLayer.h"
#include "layer/DuplicateDenseLayer.h"
#include "loss/MLE.h"
#include "loss/MPE.h"
#include "loss/MSE.h"
#include "mappings.h"
#include "misc/csv.h"
#include "misc/timer.h"
#include "network/Network.h"
#include "operations/operations.h"
#include "optimizer/Adam.h"
#include "position/fenparsing.h"
#include "position/position.h"
#include "quantitize.h"

#include <filesystem>
#include <iostream>

const std::string data_path = "E:/berserk/training-data/n5k/";
std::string output = "./resources/runs/exp18/";

float validate(Network&     network,
               DataSet&     data_set,
               DenseMatrix& target,
               SArray<bool>& target_mask,
               SparseInput& i1,
               SparseInput& i2);

int main() {
    init();

    // definitions
    constexpr uint32_t       I = 8 * 12 * 64;
    constexpr uint32_t       H = 512;
    constexpr uint32_t      L2 = 8;
    constexpr uint32_t       O = 1;
    constexpr uint32_t       B = 16384;
    constexpr uint32_t     BPE = 100000000 / B;
    constexpr  int32_t       E = 450;

    // Load files
    std::vector<std::string> files {};
    for (int i = 0; i < 20; i++)
        files.push_back(data_path + "n5k." + std::to_string(i) + ".bin");

    BatchLoader  batch_loader {files, B};
    DataSet validation = read<BINARY>(data_path + "validation.bin");

    // Input data (perspective)
    SparseInput  i0 {I, B, 32};    // 32 max inputs
    SparseInput  i1 {I, B, 32};

    DenseMatrix  target {O, B};
    SArray<bool> target_mask {O * B};
    target_mask.malloc_cpu();
    target_mask.malloc_gpu();

    // 1536 -> (2x512) -> 1
    DuplicateDenseLayer<I, H, ReLU> l1 {};
    l1.lasso_regularization = 1.0 / 8388608.0;

    DenseLayer<H * 2, L2, ReLU> l2 {};

    DenseLayer<L2, O, Sigmoid>   l3 {};
    dynamic_cast<Sigmoid*>(l3.getActivationFunction())->scalar = 1.0 / 139;

    // stack layers to build network
    std::vector<LayerInterface*> layers {};
    layers.push_back(&l1);
    layers.push_back(&l2);
    layers.push_back(&l3);

    Network network {layers};

    // loss function
    MPE     loss_function {2.5, false};
    network.setLossFunction(&loss_function);

    // optimizer
    Adam adam {};
    adam.init(layers);
    adam.alpha = 0.01;
    adam.beta1 = 0.95;
    adam.beta2 = 0.999;

    CSVWriter csv {output + "loss.csv"};
    csv.write({"epoch", "training_loss", "validation_loss"});

    Timer t {};
    for (int epoch = 1; epoch <= E; epoch++) {
        float epoch_loss = 0;
        long long prev_duration = 0;

        t.tick();

        for (int batch = 1; batch <= BPE; batch++) {
            // get the next dataset (batch)
            auto* ds = batch_loader.next();
            // assign to the inputs and compute the target
            dense_berky::assign_inputs_batch(*ds, i0, i1, target, target_mask);
            // upload relevant data
            i0.column_indices.gpu_upload();
            i1.column_indices.gpu_upload();
            target.gpu_upload();
            target_mask.gpu_upload();

            // download the loss to display the loss of the iteration
            loss_function.loss.gpu_download();

            // measure time and print output
            t.tock();
            if (batch == BPE || t.duration() - prev_duration > 1000) {
                prev_duration = t.duration();

                std::printf("\rep/ba = [%3d/%5d], ", epoch, batch + 1);
                std::printf("batch_loss = [%1.8f], ", loss_function.loss(0));
                std::printf("epoch_loss = [%1.8f], ", epoch_loss / (batch + 1));
                std::printf("speed = [%9d pos/s], ", (int) std::round(1000.0f * B * (batch + 1) / t.duration()));
                std::printf("time = [%3ds]", (int) t.duration() / 1000);
                std::cout << std::flush;
            }

            epoch_loss += loss_function.loss(0);
            // make sure to reset the loss here since the mse increments the loss in order to not have
            // to use memcpy (might change soon)
            loss_function.loss(0) = 0;
            loss_function.loss.gpu_upload();

            // feed forward
            network.batch(std::vector<SparseInput*> {&i0, &i1}, target, target_mask);

            // update weights
            adam.apply(1);
        }

        float validation_loss = validate(network, validation, target, target_mask, i0, i1);
        t.tock();
        std::printf("\rep/ba = [%3d/%5d], ", epoch, BPE);
        std::printf("valid_loss = [%1.8f], ", validation_loss);
        std::printf("epoch_loss = [%1.8f], ", epoch_loss / BPE);
        std::printf("speed = [%9d pos/s], ", (int) std::round(1000.0f * (B * BPE + validation.header.position_count) / t.duration()));
        std::printf("time = [%3ds]", (int) t.duration() / 1000);
        std::cout << std::endl;

        csv.write({std::to_string(epoch),  std::to_string(epoch_loss / BPE), std::to_string(validation_loss)});

        // computeScalars(batch_loader, network, 128, I);

        if (epoch % 10 == 0)
            write_3(output + "nn-epoch" + std::to_string(epoch) + ".nnue", network);

        if (epoch % 100 == 0)
            adam.alpha *= 0.3;
    }

    close();
}

float validate(Network&     network,
               DataSet&     data_set,
               DenseMatrix& target,
               SArray<bool>& target_mask,
               SparseInput& i1,
               SparseInput& i2) {

    int B = i1.n;

    // reset loss
    float prev_loss = network.getLossFunction()->getLoss().get(0);
    network.getLossFunction()->getLoss().get(0) = 0;
    network.getLossFunction()->getLoss().gpu_upload();

    int c = std::floor(data_set.positions.size() / B);
    for(int i = 0; i < c; i++){
        int id1 = i   * B;
        int id2 = id1 + B;
        DataSet temp{};
        temp.header.position_count = B;
        temp.positions.assign(&data_set.positions[id1],&data_set.positions[id2]);

        dense_berky::assign_inputs_batch(temp, i1, i2, target, target_mask);

        i1.column_indices.gpu_upload();
        i2.column_indices.gpu_upload();
        target.gpu_upload();
        target_mask.gpu_upload();

        network.feed(std::vector<SparseInput*> {&i1, &i2});

        network.getLossFunction()->apply(network.getOutput().values,
                                         network.getOutput().gradients,
                                         target,
                                         target_mask,
                                         DEVICE);
    }

    network.getLossFunction()->getLoss().gpu_download();
    float loss = network.getLossFunction()->getLoss().get(0);

    network.getLossFunction()->getLoss().get(0) = prev_loss;
    network.getLossFunction()->getLoss().gpu_upload();

    return loss / c;
}
