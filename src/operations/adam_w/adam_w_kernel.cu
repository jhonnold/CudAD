
#include <hip/hip_runtime.h>
// https://github.com/LiyuanLucasLiu/RAdam/blob/master/radam/radam.py#L96

__global__ void adam_w_kernel(
          float* __restrict__ values,
          float* __restrict__ gradients,
          float* __restrict__ exp_avg,
          float* __restrict__ exp_avg_sq,
          int   size,
          int   step,
          float lr,
          float beta1,
          float beta2,
          float eps,
          int   warmup) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= size) return;

    exp_avg_sq[idx] = beta2 * exp_avg_sq[idx] + (1.0 - beta2) * gradients[idx] * gradients[idx];
    exp_avg   [idx] = beta1 * exp_avg   [idx] + (1.0 - beta1) * gradients[idx];

    // we increment step in the struct, no need to do it here

    float denom = sqrtf(exp_avg_sq[idx]) + eps;
    float bc1   = 1.0 - powf(beta1, step);
    float bc2   = 1.0 - powf(beta2, step);

    float scheduled_lr = lr;
    if (warmup > step)
        scheduled_lr = 1e-8 + step * lr / warmup;

    float step_size = scheduled_lr * sqrtf(bc2) / bc1;
    float delta     = step_size * exp_avg[idx] / denom;

    values[idx]   -= delta;
    gradients[idx] = 0;
}