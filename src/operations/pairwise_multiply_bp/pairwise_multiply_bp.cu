#include "hip/hip_runtime.h"
/**
    CudAD is a CUDA neural network trainer, specific for chess engines.
    Copyright (C) 2022 Finn Eggers

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <https://www.gnu.org/licenses/>.
 */

#include "pairwise_multiply_bp.h"

// clang-format off
__global__ void pairwise_multiply_bp_kernel(
    const float* __restrict__ input,
          float* __restrict__ input_grd,
    const float* __restrict__ output_grd,
    unsigned int outsize,
    unsigned int neurons){
    // clang-format on

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= outsize)
        return;

    int halfsize = neurons / 2;

    int idx1 = idx + halfsize * (idx / halfsize);
    int idx2 = idx1 + halfsize;

    input_grd[idx1] = output_grd[idx] * input[idx2];
    input_grd[idx2] = output_grd[idx] * input[idx1];
}