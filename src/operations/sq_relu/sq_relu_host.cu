#include "hip/hip_runtime.h"
/**
    CudAD is a CUDA neural network trainer, specific for chess engines.
    Copyright (C) 2022 Finn Eggers

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <https://www.gnu.org/licenses/>.
 */

#include "sq_relu.h"

// clang-format off
void sq_relu_host(
    const float* A,
          float* B,
    unsigned int size){
    // clang-format on

    for (int i = 0; i < size; i++) {
        B[i] = std::max(A[i], 0.0f) * A[i];
    }
}
