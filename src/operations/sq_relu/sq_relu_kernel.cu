#include "hip/hip_runtime.h"
/**
    CudAD is a CUDA neural network trainer, specific for chess engines.
    Copyright (C) 2022 Finn Eggers

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <https://www.gnu.org/licenses/>.
 */

#include "sq_relu.h"

// clang-format off
__global__ void sq_relu_kernel(
    const float* __restrict__ A,
          float* __restrict__ B,
    unsigned int size){
    // clang-format on

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= size)
        return;

    B[idx] = max(0.0f, A[idx]) * A[idx];
}
