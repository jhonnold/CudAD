/**
    CudAD is a CUDA neural network trainer, specific for chess engines.
    Copyright (C) 2022 Finn Eggers

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <https://www.gnu.org/licenses/>.
 */

#include "sq_relu_bp.h"

#include <iostream>

// clang-format off
void sq_relu_bp_host(
    const float* A,
          float* A_grd,
    const float* B,
    const float* B_grd,
    unsigned int size){
    // clang-format on

    for (int idx = 0; idx < size; idx++) {
        if (B[idx] > 0) {
            A_grd[idx] = 2 * A[idx] * B_grd[idx];
        } else {
            A_grd[idx] = 0;
        }
    }
}
