#include "hip/hip_runtime.h"
/**
    CudAD is a CUDA neural network trainer, specific for chess engines.
    Copyright (C) 2022 Finn Eggers

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <https://www.gnu.org/licenses/>.
 */

#include "sq_relu_bp.h"

// clang-format off
__global__ void sq_relu_bp_kernel(
    const float* __restrict__ A,
          float* __restrict__ A_grd,
    const float* __restrict__ B,
    const float* __restrict__ B_grd,
    unsigned int size){
    // clang-format on

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= size)
        return;

    if (B[idx] > 0) {
        A_grd[idx] = 2 * A[idx] * B_grd[idx];
    } else {
        A_grd[idx] = 0;
    }
}
